
#include <hip/hip_runtime.h>
#include <stdio.h>

#define WARP_SIZE 32

#define GROUPSET 16
#define NUMFACES 3

#define fouralpha 1.82
#define fouralpha4 5.82


#define Connect(a,b,c) Connect[ a + 3 * ( b + mC * c ) ]

__device__ __forceinline__ double shfl_d(double var,int lane)
{ float lo, hi;
  asm volatile("mov.b64 {%0,%1}, %2;" : "=f"(lo), "=f"(hi) : "d"(var));
  hi = __shfl(hi, lane);
  lo = __shfl(lo, lane);
  asm volatile("mov.b64 %0, {%1,%2};" : "=d"(var) : "f"(lo), "f"(hi));
  return var;
}


extern "C"
{


  __global__ void GPU_sweep(
			    int  size_maxCorner,
			    int  size_maxcf,
			    int  nAngle,
			    int  nzones,
			    int  ncornr,
			    int  Groups,
			    int  nbelem,
			    int* AngleOrder,
			    double* soa_omega,
			    int* nextZ,
			    int* next,
			    int* soa_nCorner,
			    int* soa_nCFaces,
			    int* soa_c0,
			    double* soa_STotal,
			    double* soa_STime,
			    double* soa_SigtInv,
			    double* soa_Volume,
			    double* soa_Sigt,
			    double* soa_A_fp,
			    double* soa_A_ez,
			    int* soa_Connect,
			    double* psic,
			    double* psib,
			    double* omega_A_fp,
			    double* omega_A_ez,
			    int* soa_Connect_ro,
			    int* passZ)
  {

    //   double omega[3];
    int c,ig,i,icface,ifp,cez,k;
    //   double Q[Groups * size_maxCorner];
    //   double src[Groups * size_maxCorner];
    //   double SigtVol[Groups * size_maxCorner];
    //   double afpm[size_maxcf];
    //   double psifp[Groups * size_maxcf];
    //   int    ez_exit[size_maxcf];
    //   double coefpsic[size_maxcf];
    //   double tpsic[Groups * size_maxCorner];
    //   double psi_opp[Groups];
    double area_opp,area_inv,sumArea;
    double r_psifp;

    double psi_opp,tpsic,r_afpm;
    double Q[8];
    double src[8];

    //double volume[8];
    //double coefpsic_stk[3];
    //double psifp[3];
    //int ez_exit[3];
  
    //double *src;
    volatile double *volume;
    volatile double *coefpsic;
    volatile double *psifp;
    volatile int *ez_exit;
    __shared__ volatile double sm_agg[12*128];  // 4x32 thread per tb. 8tb. 6KB
   
    int offset = (8+3+96+3)*threadIdx.y;
    volume = &(sm_agg[offset]);  //8 doubles  
    offset += size_maxCorner;

    coefpsic = &(sm_agg[offset]); // 3 doubles
    offset += size_maxcf;

    psifp = &(sm_agg[offset]); // 3 x 32 doubles
    offset += size_maxcf * WARP_SIZE;

    //note ez_exit has integer type
    ez_exit = (int*) &(sm_agg[offset]); // 3 int
    //   for(int Angle=0;Angle<nAngle;Angle++)

    //   const double fouralpha = 1.82;
    //   const double fouralpha4 = 5.82;
   
#define soa_omega(a,b) soa_omega[a + 3 * b]

    //   #define tpsic(ig,c) tpsic[ (ig) + Groups * (c)]
#define EB_ListExit(a,ia) EB_ListExit[ a + 2 * (ia) ]
#define soa_A_fp(a,icface,c,zone) soa_A_fp[ a + 3 * ( icface + size_maxcf * ( c + size_maxCorner * (zone) ) )]
#define soa_A_ez(a,icface,c,zone) soa_A_ez[ a + 3 * ( icface + size_maxcf * ( c + size_maxCorner * (zone) ) )]
#define omega_A_fp(icface,c,zone) omega_A_fp[ ( icface + size_maxcf * ( c + size_maxCorner * (zone) ) )]
#define omega_A_ez(icface,c,zone) omega_A_ez[ ( icface + size_maxcf * ( c + size_maxCorner * (zone) ) )]
#define soa_Connect(a,icface,c,zone) soa_Connect[ a + 3 * ( icface + size_maxcf * ( c + size_maxCorner * (zone) ) )]
   
#define psifp(ig,jf) psifp[(ig) + WARP_SIZE * (jf)]
#define psib(ig,b,c) psib[(ig) + Groups * ((b) + nbelem * (c) )]
#define psic(ig,b,c) psic[(ig) + Groups * ((b) + ncornr *(c) )]
   
#define Q(ig,c) Q[(ig) + WARP_SIZE * (c)]
    //#define src(ig,c) src[(ig) + Groups * (c)]
#define src(ig,c) src[c]
    //  #define SigtVol(ig,c) SigtVol[(ig) + Groups * (c)]
#define soa_Sigt(ig,zone) soa_Sigt[(ig) + Groups * (zone)]
#define soa_Volume(c,zone) soa_Volume[c + size_maxCorner * (zone)]
#define soa_SigtInv(ig,zone) soa_SigtInv[(ig) + Groups * (zone)]
#define soa_STotal(ig,c,zone) soa_STotal[ig + Groups * ( c + size_maxCorner * (zone) )]
//#define soa_STime(ig,c,Angle,zone) soa_STime[ig + Groups * ( c + size_maxCorner * ( Angle + nAngle * (zone) ) )]
#define soa_STime(ig,ic,Angle) soa_STime[ig + Groups * ( (ic) + ncornr * (Angle) ) ]
#define nextZ(a,b) nextZ[ (a) + nzones * (b) ]
#define next(a,b) next[ (a) + (ncornr+1)  * (b) ]

    //int mm = blockIdx.x;
    int Angle = AngleOrder[blockIdx.x]-1;
    ig = threadIdx.x;
 

    //   if(ig==0) printf("my offset=%d\n",offset);
    //   if(ig==0)
    //   {
    //     printf("psic=%x\n",psic);
    //     printf("nextZ=%x\n",psic);
    //     printf("next=%x\n",psic);
    //     printf("psib=%x\n",psic);
    //   }


    omega_A_fp += Angle * nzones * size_maxcf * size_maxCorner;
    omega_A_ez += Angle * nzones * size_maxcf * size_maxCorner;
    passZ      += Angle * nzones;
   
    const int group_offset=blockIdx.y * 32;

    //  if (!( group_offset + threadIdx.x < Groups )) return;
    
    psib += group_offset;
    psic += group_offset;
    soa_Sigt += group_offset;
    soa_STotal += group_offset;
    soa_SigtInv += group_offset;
    soa_STime += group_offset;

    int ndone = 0;
    int ndoneZ = 0;
    // hyperplane number p
    int p=0;

    while(ndoneZ < nzones)
    {
      //increment hyperplane
      p++;
      // get number of zones in this hyperplane
      int passZcnt = passZ[p] - passZ[p-1];

      for(int ii=threadIdx.y;ii<passZcnt;ii+=blockDim.y)
      {
	ndone = ( ndoneZ + ii ) * size_maxCorner;
 
	// get the zone (minus 1 so it is valid c index)
	int zone = nextZ(ndoneZ+ii,Angle) - 1;

	//       if(ig==0 && blockIdx.x==0) printf("ang=%d zone=%d tidy=%d ndoneZ=%d\n",blockIdx.x,zone,threadIdx.y,ndoneZ);
  
  
	int nCorner   = soa_nCorner[zone];
	int nCFaces   = soa_nCFaces[zone];
	int c0        = soa_c0[zone] ;
  
	double Sigt = soa_Sigt(ig,zone);
	double r_soa_SightInv = soa_SigtInv(ig,zone);
	double r_omega_A_fp;
	double r_omega_A_ez;
	int connect0,connect1,connect2;
  
	if(ig<nCorner) volume[ig] = soa_Volume(ig,zone);

	if(ig<nCorner*nCFaces)
	{
          int cc = size_maxcf * size_maxCorner;
          r_omega_A_fp = omega_A_fp[ig + cc * zone];
          r_omega_A_ez = omega_A_ez[ig + cc * zone];
          connect0     = soa_Connect_ro[ig + cc*(0 + 3*zone)];
          connect1     = soa_Connect_ro[ig + cc*(1 + 3*zone)];
          connect2     = soa_Connect_ro[ig + cc*(2 + 3*zone)];
	}
   
  
	for(c=0;c<nCorner;c++)
	{
	  double source = soa_STotal(ig,c,zone) + soa_STime(ig,c0+c,Angle);
	  Q[c]       = r_soa_SightInv *source ;
	  //src(ig,c)     = soa_Volume(c,zone) *source;
	  //volume[c] = soa_Volume(c,zone);
	  src(ig,c)     = volume[c]*source;
	  //SigtVol(ig,c) = soa_Sigt(ig,zone)*soa_Volume(c,zone);
	}
  
	for(i=0;i<nCorner;i++)
	{
  
	  int ic      = next(ndone+i,Angle);
	  c       = ic - c0 - 1;
  
	  sumArea = 0.0;
   
	  for(icface=0;icface<nCFaces;icface++)
	  {
	    //afpm[icface] = omega_A_fp(icface,c,zone);  
	    r_afpm = shfl_d(r_omega_A_fp,icface+size_maxcf*c);
  
	    //  if ( Angle == 1 && ig==0 && zone == 1 )
	    //    printf("a=%d,c=%d,icface=%d,afpm=%e\n",Angle,c,icface,r_afpm);
  
	    //       int icfp    = soa_Connect(0,icface,c,zone) - 1;
	    //       int ib      = soa_Connect(1,icface,c,zone) - 1;
	    int icfp= __shfl(connect0,icface+size_maxcf*c);
	    int ib= __shfl(connect1,icface+size_maxcf*c);
  
	    //         if ( Angle == 1 && ig==0 && zone == 1 )
	    //               printf("a=%d,c=%d,icface=%d,afpm=%e\n",Angle,c,icface,r_afpm);
                                                                                                     
	    if ( r_afpm >= 0.0 )
	    { 
	      sumArea = sumArea + r_afpm;
	    }
	    else
	    {
	      if (icfp == -1)
	      {
		//             psifp(ig,icface) = psib(ig,ib,Angle);
		r_psifp = psib(ig,ib,Angle);
		//    if ( Angle == 1 && ig==0 && zone == 0 )
		//      printf("a=%d,c=%d,icface=%d,zone=%d,icfp=%d,ib=%d,%e\n",Angle,c,icface,zone,icfp,ib,r_psifp);
	      }
	      else
	      {
		//             psifp(ig,icface) = psic(ig,icfp,Angle);
		//             printf("psic(%d,%d,%d)\n",ig,icfp,Angle);
		r_psifp = psic(ig,icfp,blockIdx.x);
		//          if ( Angle == 1 && ig==0 && zone == 0 )
		//            printf("a=%d,c=%d,icface=%d,zone=%d,icfp=%d,ib=%d,%e\n",Angle,c,icface,zone,icfp,ib,r_psifp);
            
	      }
  
	      src(ig,c)  -= r_afpm*r_psifp;
	      psifp(ig,icface) = r_psifp;
	      //psifp[icface] = r_psifp;
	    }
	    //       if ( Angle == 1 && ig==0 && zone == 1 )
	    //              printf("a=%d,c=%d,icface=%d,afpm=%e\n",Angle,c,icface,afpm[icface]);
	  }
  
  
	  //       if ( Angle == 1 && ig < 5 && c == 2 && zone == 1 )
	  //         printf("a=%d,g=%d,c=%d,psifp=%e,sumArea=%e\n",Angle,ig,c,psifp(ig,1),sumArea);
  
  
	  int nxez = 0;
  
	  for(icface=0;icface<nCFaces;icface++)
	  {
  
	    //double aez = omega_A_ez(icface,c,zone);
	    double aez = shfl_d(r_omega_A_ez,icface+size_maxcf*c);
	    //                     if ( Angle == 1 && ig==0 && zone == 0 )
	    //                            printf("a=%d,c=%d,aez=%e,icface=%d\n",Angle,c,aez,icface);
  
	    if (aez > 0.0 )
	    {
  
	      sumArea        = sumArea + aez;
	      area_opp       = .0;
	      //           cez            = soa_Connect(2,icface,c,zone) - 1;
	      cez            = __shfl(connect2,icface+size_maxcf*c);
	      ez_exit[nxez]  = cez;
	      coefpsic[nxez] = aez;
	      nxez           = nxez + 1;
  
	      if (nCFaces == 3)
	      {
  
		ifp = (icface+1)%nCFaces;
		r_afpm = shfl_d(r_omega_A_fp,ifp+size_maxcf*c);
                //       if ( Angle == 1 && ig==0 && zone == 1 )
                //         printf("a=%d,c=%d,ifp=%d,afpm=%e\n",Angle,c,ifp,afpm[ifp]);
  
		if ( r_afpm < 0.0 )
		{ 
		  area_opp   = -r_afpm;
		  psi_opp =  psifp(ig,ifp);
		  //psi_opp =  psifp[ifp];
		}
	      }
	      else
	      {
  
		ifp        = icface;
		area_opp   = 0.0;
		psi_opp = 0.0;
  
		for(k=0;k<nCFaces-2;k++)
		{
		  ifp = (ifp+1)%nCFaces;
		  r_afpm = shfl_d(r_omega_A_fp,ifp+size_maxcf*c);
		  if ( r_afpm < 0.0 )
		  {
		    area_opp   = area_opp   - r_afpm;
		    psi_opp = psi_opp - r_afpm*psifp(ig,ifp);
		    //psi_opp = psi_opp - r_afpm*psifp[ifp];
		  }
		}
  
		area_inv = 1.0/area_opp;
  
		psi_opp = psi_opp*area_inv;
  
	      }
  
	      if (area_opp > 0.0) {
  
		double aez2 = aez*aez;
  
		{
    
		  double sigv         = Sigt*volume[c];
		  double sigv2        = sigv*sigv;
		  double gnum         = aez2*( fouralpha*sigv2 +       aez*(4.0*sigv + 3.0*aez) );
		  double gtau         = gnum/( gnum + 4.0*sigv2*sigv2 + aez*sigv*(6.0*sigv2 + 2.0*aez*(2.0*sigv + aez)) ) ;
		  double sez          = gtau*sigv*( psi_opp - Q[c] ) +   0.5*aez*(1.0 - gtau)*( Q[c] - Q[cez] );
  
		  src(ig,c)    = src(ig,c)   + sez;
		  src(ig,cez)  = src(ig,cez) - sez;
  
		  //                       if ( Angle == 1 && ig < 5 && zone == 0 )
		  //                                    printf("a=%d,g=%d,c=%d,cez=%d,icface=%d,src(c)=%e,src(cez)=%e,sez=%e\n",Angle,ig,c,cez,icface,src(ig,c),src(ig,cez),sez);
  
		}
  
	      }
	      else
	      {
		double sez          = 0.5*aez*( Q[c] - Q[cez] );
		src(ig,c)    = src(ig,c)   + sez;
		src(ig,cez)  = src(ig,cez) - sez;
  
	      } 
	    }
	  }
  
	  //       printf("ckim angle,zone,corner,aez_cnt %d,%d,%d,%d\n",Angle,zone,c,aez_cnt);
  
  
	  tpsic = src(ig,c)/(sumArea + Sigt*volume[c]);
  
  
	  for(icface=0;icface<nxez;icface++)
	  {
	    int cez   = ez_exit[icface];
	    src(ig,cez) = src(ig,cez) + coefpsic[icface]*tpsic;
	  }
  
	  //hope that ther is no self referencing
	  psic(ig,c0+c,blockIdx.x) = tpsic;
	  //psibatch(ig,c0+c,mm)= tpsic;

	  //     if ( Angle == 1 && ig < 5 && zone == 0 )
	  //                  printf("a=%d,g=%d,c=%d,psic=%e,corner=%d\n",Angle,ig,c,tpsic,c0+c);
	} //end of corner 
  
      } //end of zone loop 
      ndoneZ += passZcnt;
      __syncthreads();
    } //end of while

  }


__global__ void GPU_fp_ez(
          int  size_maxCorner,
          int  size_maxcf,
          int  nAngle,
          int  nzones,
          int  ncornr,
          int  Groups,
          int  nbelem,
          int* AngleOrder,
       double* soa_omega,
          int* nextZ,
          int* next,
          int* soa_nCorner,
          int* soa_nCFaces,
          int* soa_c0,
       double* soa_A_fp,
       double* soa_A_ez,
       double* omega_A_fp,
       double* omega_A_ez,
          int* soa_Connect,
          int* soa_Connect_ro)
  {

//   double omega[3];
   int c,i,ig,icface,ii;
//   double Q[Groups * size_maxCorner];
//   double src[Groups * size_maxCorner];
//   double SigtVol[Groups * size_maxCorner];
//   double afpm[size_maxcf];
//   double psifp[Groups * size_maxcf];
//   int    ez_exit[size_maxcf];
//   double coefpsic[size_maxcf];
//   double tpsic[Groups * size_maxCorner];
//   double psi_opp[Groups];

   double omega0, omega1, omega2;
   

   
//   const double fouralpha4 = 5.82;
   
   #define soa_omega(a,b) soa_omega[a + 3 * b]
   #define omega_A_fp(icface,c,zone) omega_A_fp[  ( icface + size_maxcf * ( c + size_maxCorner * (zone) ) )]
   #define omega_A_ez(icface,c,zone) omega_A_ez[  ( icface + size_maxcf * ( c + size_maxCorner * (zone) ) )]

//   #define tpsic(ig,c) tpsic[ (ig) + Groups * (c)]
   #define EB_ListExit(a,ia) EB_ListExit[ a + 2 * (ia) ]
   #define soa_A_fp(a,icface,c,zone) soa_A_fp[ a + 3 * ( icface + size_maxcf * ( c + size_maxCorner * (zone) ) )]
   #define soa_A_ez(a,icface,c,zone) soa_A_ez[ a + 3 * ( icface + size_maxcf * ( c + size_maxCorner * (zone) ) )]
   #define soa_Connect(a,icface,c,zone) soa_Connect[ a + 3 * ( icface + size_maxcf * ( c + size_maxCorner * (zone) ) )]
   #define soa_Connect_ro(a,icface,c,zone) soa_Connect_ro[ icface + size_maxcf * ( c + size_maxCorner * ( a + 3 * zone) ) ]
   
   //#define psifp(ig,jf) psifp[(ig) + Groups * (jf)]
   #define psib(ig,b,c) psib[(ig) + Groups * ((b) + nbelem * (c) )]
   
     //#define Q(ig,c) Q[(ig) + Groups * (c)]
     //#define src(ig,c) src[(ig) + Groups * (c)]
//  #define SigtVol(ig,c) SigtVol[(ig) + Groups * (c)]
   #define soa_Sigt(ig,zone) soa_Sigt[(ig) + Groups * (zone)]
   #define soa_SigtInv(ig,zone) soa_SigtInv[(ig) + Groups * (zone)]
   #define soa_STotal(ig,c,zone) soa_STotal[ig + Groups * ( c + size_maxCorner * (zone) )]
//   #define soa_STime(ig,c,Angle,zone) soa_STime[ig + Groups * ( c + size_maxCorner * ( Angle + nAngle * (zone) ) )]
   #define nextZ(a,b) nextZ[ (a) + nzones * (b) ]
   #define next(a,b) next[ (a) + (ncornr+1)  * (b) ]





//   for(int Angle=0;Angle<nAngle;Angle++)

   int Angle = 32*blockIdx.x + threadIdx.x;

   omega0 = soa_omega(0,Angle);
   omega1 = soa_omega(1,Angle);
   omega2 = soa_omega(2,Angle);

   int ndone = 0;

   omega_A_fp += Angle * nzones * size_maxcf * size_maxCorner;
   omega_A_ez += Angle * nzones * size_maxcf * size_maxCorner;

   for(ii=0;ii<nzones;ii++)
   {
 
     int zone = nextZ(ii,Angle) - 1;


     int nCorner   = soa_nCorner[zone];
     int nCFaces   = soa_nCFaces[zone];
     int c0        = soa_c0[zone] ;

     for(i=0;i<nCorner;i++)
     {

       int ic      = next(ndone+i,Angle);
       c       = ic - c0 - 1;

 
       for(icface=0;icface<nCFaces;icface++)
       {
         omega_A_fp(icface,c,zone) =  omega0*soa_A_fp(0,icface,c,zone) + 
                        omega1*soa_A_fp(1,icface,c,zone) + 
                        omega2*soa_A_fp(2,icface,c,zone);
         int icfp    = soa_Connect(0,icface,c,zone) - 1;
         int ib      = soa_Connect(1,icface,c,zone) - 1;
         int cez     = soa_Connect(2,icface,c,zone) - 1;
         soa_Connect_ro(0,icface,c,zone) = icfp;
         soa_Connect_ro(1,icface,c,zone) = ib  ;
         soa_Connect_ro(2,icface,c,zone) = cez ;

         //if (ig==0) printf("Angle,zone,c,icface,afp=%d,%d,%d,%d,%f\n",Angle,zone,c,icface,omega_A_fp(icface,c,zone));
       }


       for(icface=0;icface<nCFaces;icface++)
       {

         omega_A_ez(icface,c,zone) = omega0*soa_A_ez(0,icface,c,zone) + omega1*soa_A_ez(1,icface,c,zone) + omega2*soa_A_ez(2,icface,c,zone) ;
         //if (ig==0) printf("Angle,zone,c,icface,afp=%d,%d,%d,%d,%f\n",Angle,zone,c,icface,omega_A_ez(icface,c,zone));
       }


     } 

     ndone = ndone + nCorner;
   }
  }




}






